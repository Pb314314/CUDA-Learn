#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vector_add(const float* A, const float* B, float* C, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < N){
        C[index] = A[index] + B[index];
    }
}

int main(){
    int N = 32 * 1024 * 1024;
    size_t size = N * sizeof(float);
    
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    for (int i=0;i<N;i++){
        h_A[i] = i;
        h_B[i] = 2 * i;
    } 

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 Grid(N / 256);
    dim3 Block(64);

    vector_add<<<Grid, Block>>>(d_A, d_B, d_C, size);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    return 0;
}